
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MIN_PRINTABLE 32
#define MAX_PRINTABLE 127
#define PRINTABLE_RANGE (MAX_PRINTABLE - MIN_PRINTABLE + 1)
#define OFFSET 5

#define TOTALTHREADS 1024
#define THREADS_IN_BLOCK 128
__constant__ char const_in_text[TOTALTHREADS];


__host__ hipEvent_t get_time(void) {
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
    return time;
}

/**
 * Generates an array of random characters
 */
void fillRandArray(char *input_text, int totalThreads) {
   for (int i = 0; i < TOTALTHREADS; i++)
    {
        int val = rand() % (PRINTABLE_RANGE);              
        input_text[i] = (char) val + MIN_PRINTABLE;                          
    }
}

/**
 * Allocates memory for hosts input and output arrays.
 * Initializes the input array with random characters.                                
 */

 void Alloc(char **input_text, char **result) {
                    
    // allocate host memory
    char *in, *out;
    in = (char*)malloc(TOTALTHREADS*sizeof(char));
    out = (char*)malloc(TOTALTHREADS*sizeof(char));
   
    // populate input array
    fillRandArray(in, TOTALTHREADS);

    // update pointers                           
    *input_text = in;
    *result = out;
}

// ******************************** CONSTANT ******************************* //                                

/**
 * Perform Caesar cipher on an array of characters in parallel.
 * Passing in -OFFSET reverses the operation.
 */
 __global__ void constant_encrypt(char *result) { 
    
    // Calculate the current index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
     
	/* 
	 * Adjust value of text and key to be based at 0 
	 * Printable ASCII starts at MIN_PRINTABLE, but 0 start is easier to work with 
	 */
    int ascii = const_in_text[idx];
    if (ascii < 32 || ascii > 127)
        printf("Enountered character outside of printable range");
    
	int zeroed_ascii = ascii - MIN_PRINTABLE;
    signed int offset = OFFSET;
    
	// Encrypt by adding the offset value and taking mod to wrap
    int tmp = (zeroed_ascii + offset) % (PRINTABLE_RANGE);
    
    // Handle negative operands..
    int cipherchar = tmp < 0 ? (tmp + PRINTABLE_RANGE) : tmp;
	cipherchar += MIN_PRINTABLE;
	result[idx] = cipherchar;
}

float const_gpu_cipher(int numBlocks, char *input_text, char *result, 
                        char *gpu_out) {

    // copy data from host to gpu
    hipMemcpyToSymbol(HIP_SYMBOL(const_in_text), input_text, TOTALTHREADS * sizeof(char));

     // Begin timing
    hipEvent_t start_time = get_time();
                            
    // compute results on gpu
    constant_encrypt<<<numBlocks, TOTALTHREADS/numBlocks>>>(gpu_out);
    
    // End timing
    hipEvent_t end_time = get_time();
    hipEventSynchronize(end_time);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start_time, end_time);
    
    // copy back to cpu 
    hipMemcpy(result, gpu_out, TOTALTHREADS * sizeof(char), 
    hipMemcpyDeviceToHost);
    
    hipEventDestroy(start_time);
    hipEventDestroy(end_time);    
                                    
    return elapsed;
}
/**
 * Allocates memory, calls cipher, and cleans up. Input array is stored
 * in constant memory.                             
 */
void const_main(int numBlocks){

    // Initialize input array with random characters
    char *input_text, *result;
    Alloc(&input_text, &result);
    
    // Allocate gpu memory. Don't need malloc gpu_in for constant memory
    char *gpu_out;
    hipMalloc((void**)&gpu_out, TOTALTHREADS * sizeof(char));

    // Perform encryption
    float elapsed;
    elapsed = const_gpu_cipher(numBlocks,
                    input_text, result, gpu_out);
                        
    // clean up 
    hipFree(gpu_out);
    free(input_text);
    free(result);

    printf("Constant memory elapsed: %3.3f ms\n", elapsed);                           
}

// ******************************** SHARED ******************************* //

/**
 * Perform Caesar cipher on an array of characters in parallel.
 * Passing in -OFFSET reverses the operation.
 */
 __global__ void shared_encrypt(char *input_text, char *result) { 
    
    __shared__ char shared_in_text[THREADS_IN_BLOCK];

    // Calculate the current index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int idx_in_block = threadIdx.x;
    shared_in_text[idx_in_block] = input_text[idx];

	/* 
	 * Adjust value of text and key to be based at 0 
	 * Printable ASCII starts at MIN_PRINTABLE, but 0 start is easier to work with 
	 */ 
    int ascii = shared_in_text[idx_in_block];
    if (ascii < 32 || ascii > 127)
        printf("Enountered character outside of printable range");
    
	int zeroed_ascii = ascii - MIN_PRINTABLE;
    signed int offset = OFFSET;
    
	// Encrypt by adding the offset value and taking mod to wrap
    int tmp = (zeroed_ascii + offset) % (PRINTABLE_RANGE);
    
    // Handle negative operands..
    int cipherchar = tmp < 0 ? (tmp + PRINTABLE_RANGE) : tmp;
	cipherchar += MIN_PRINTABLE;
	result[idx] = cipherchar;
}

/**
 * Calls cipher kernel and executes on gpu. Host -> device memory transfer
 * is timed. Shared memory is used for the input array.                  
 */                                 
 float shared_gpu_cipher(int numBlocks, int totalThreads, char *input_text,
    char *result, char *gpu_in, char *gpu_out) {

    // copy data from host to gpu

    hipMemcpy(gpu_in, input_text, totalThreads * sizeof(char), 
    hipMemcpyHostToDevice);
    
     // Begin timing
    hipEvent_t start_time = get_time();
    
    // compute results on gpu
    shared_encrypt<<<numBlocks, totalThreads/numBlocks>>>(gpu_in, gpu_out);
    
     // End timing
    hipEvent_t end_time = get_time();
    hipEventSynchronize(end_time);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start_time, end_time);
    
    // copy back to cpu 
    hipMemcpy(input_text, gpu_in, totalThreads * sizeof(char), 
    hipMemcpyDeviceToHost);
    hipMemcpy(result, gpu_out, totalThreads * sizeof(char), 
    hipMemcpyDeviceToHost);
    
    hipEventDestroy(start_time);
    hipEventDestroy(end_time); 
                                  
    return elapsed;
} 
/**
 * Allocates pinned memory, calls cipher, and cleans up                             
 */
void shared_main(int numBlocks, int totalThreads) {

    // Initialize input array with random characters
    char *input_text, *result;
    Alloc(&input_text, &result);

    // Allocate gpu memory  
    char *gpu_in, *gpu_out;
    hipMalloc((void**)&gpu_in, totalThreads * sizeof(char));
    hipMalloc((void**)&gpu_out, totalThreads * sizeof(char));

    // Perform encryption
    float elapsed;
    elapsed = shared_gpu_cipher(numBlocks, totalThreads,
                input_text, result, gpu_in, gpu_out);
                    
    // clean up                           
    hipFree(gpu_in);
    hipFree(gpu_out);
    hipFree(input_text);
    hipFree(result);
    
    printf("Shared memory elapsed: %3.3f ms\n", elapsed);                           
}
       
int main(int argc, char** argv) {

	int numBlocks = TOTALTHREADS/THREADS_IN_BLOCK;
    
	// validate command line arguments
	if (TOTALTHREADS % THREADS_IN_BLOCK != 0) {

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("Please update and re-rerun \n");
    }
    
    const_main(numBlocks); 
    shared_main(numBlocks, TOTALTHREADS);

	return EXIT_SUCCESS;
}