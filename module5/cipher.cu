
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MIN_PRINTABLE 32
#define MAX_PRINTABLE 127
#define PRINTABLE_RANGE (MAX_PRINTABLE - MIN_PRINTABLE) + 1
#define OFFSET 5

/* forward declaration */
float gpu_cipher(int numBlocks, int totalThreads, char *input_text,
    char *result, char *gpu_in, char *gpu_out);

__host__ hipEvent_t get_time(void) {
    hipEvent_t time;
    hipEventCreate(&time);
    hipEventRecord(time);
    return time;
}
                
/**
 * Perform Caesar cipher on an array of characters in parallel.
 * Passing in -OFFSET reverses the operation.
 */
__global__ void encrypt(char *input_text, char *result) { 
    
    // Calculate the current index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
     
	/* 
	 * Adjust value of text and key to be based at 0 
	 * Printable ASCII starts at MIN_PRINTABLE, but 0 start is easier to work with 
	 */ 
    int ascii = input_text[idx];
    if (ascii < 32 || ascii > 127)
        printf("Enountered character outside of printable range");
    
	int zeroed_ascii = ascii - MIN_PRINTABLE;
    signed int offset = OFFSET;
    
	// Encrypt by adding the offset value and taking mod to wrap
    int tmp = (zeroed_ascii + offset) % (PRINTABLE_RANGE);
    
    // Handle negative operands..
    int cipherchar = tmp < 0 ? (tmp + PRINTABLE_RANGE) : tmp;
	cipherchar += MIN_PRINTABLE;
	result[idx] = cipherchar;
}
    

/**
 * Generates an array of random characters
 */
void fillRandArray(char *input_text, int totalThreads) {
   for (int i = 0; i < totalThreads; i++)
    {
        int val = rand() % (PRINTABLE_RANGE);              
        input_text[i] = (char) val + MIN_PRINTABLE;                          
    }
}

// ******************************** PAGEABLE ******************************* //                                
/**
 * Allocates pageable memory for hosts input and output arrays.
 * Initializes the input array with random characters.                                
 */
void pageableAlloc(int totalThreads, char **input_text,
                char **result) {
                    
    // allocate host memory
    char *in, *out;
    in = (char*)malloc(totalThreads*sizeof(char));
    out = (char*)malloc(totalThreads*sizeof(char));
   
    // populate input array
    fillRandArray(in, totalThreads);
                                    
    // update pointers                           
    *input_text = in;
    *result = out;
}
    
/**
 * Allocates pageable memory, calls cipher, and cleans up                             
 */
void pageable_main(int numBlocks, int totalThreads){

    // Initialize input array with random characters
    char *input_text, *result;
    pageableAlloc(totalThreads, &input_text, &result);
    
    // Allocate gpu memory  
    char *gpu_in, *gpu_out;
    hipMalloc((void**)&gpu_in, totalThreads * sizeof(char));
    hipMalloc((void**)&gpu_out, totalThreads * sizeof(char));

    // Perform encryption
    float elapsed;
    elapsed = gpu_cipher(numBlocks, totalThreads,
                    input_text, result, gpu_in, gpu_out);
                        
    // clean up                           
    hipFree(gpu_in);
    hipFree(gpu_out);
    free(input_text);
    free(result);

    printf("Host -> device transfer with pageable mem: %3.3f ms\n", elapsed);                           
}

// ******************************** PINNED ******************************* //
/**
 * Allocates pinned memory for hosts input and output arrays.
 * Initializes the input array with random characters.                                
 */
void pinnedAlloc(int totalThreads, char **input_text,
    char **result) {

    // allocate
    char *in, *out;
    hipHostAlloc((void**)&in,
                        totalThreads*sizeof(char),
                        hipHostMallocDefault);
    hipHostAlloc((void**)&out,
                        totalThreads*sizeof(char),
                        hipHostMallocDefault);

    // populate input array
    fillRandArray(in, totalThreads);
                            
    // update pointers                           
    *input_text = in;
    *result = out;
}
    
/**
 * Allocates pinned memory, calls cipher, and cleans up                             
 */
void pinned_main(int numBlocks, int totalThreads) {

    // Initialize input array with random characters
    char *input_text, *result;
    pinnedAlloc(totalThreads, &input_text, &result);

    // Allocate gpu memory  
    char *gpu_in, *gpu_out;
    hipMalloc((void**)&gpu_in, totalThreads * sizeof(char));
    hipMalloc((void**)&gpu_out, totalThreads * sizeof(char));

    // Perform encryption
    float elapsed;
    elapsed = gpu_cipher(numBlocks, totalThreads,
                input_text, result, gpu_in, gpu_out);
                    
    // clean up                           
    hipFree(gpu_in);
    hipFree(gpu_out);
    hipFree(input_text);
    hipFree(result);

    printf("Host -> device transfer with pinned mem: %3.3f ms\n", elapsed);                           
}
    
/**
 * Calls cipher kernel and executes on gpu. Host -> device memory transfer
 * is timed.                             
 */                                 
float gpu_cipher(int numBlocks, int totalThreads, char *input_text,
    char *result, char *gpu_in, char *gpu_out) {

    // Begin timing
    hipEvent_t start_time = get_time();

    // copy data from host to gpu
    hipMemcpy(gpu_in, input_text, totalThreads * sizeof(char), 
    hipMemcpyHostToDevice);
    hipMemcpy(gpu_out, result, totalThreads * sizeof(char), 
    hipMemcpyHostToDevice);

    // End timing
    hipEvent_t end_time = get_time();
    hipEventSynchronize(end_time);
    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start_time, end_time);

    // compute results on gpu
    encrypt<<<numBlocks, totalThreads/numBlocks>>>(gpu_in, gpu_out);
    
    // copy back to cpu 
    hipMemcpy(input_text, gpu_in, totalThreads * sizeof(char), 
    hipMemcpyDeviceToHost);
    hipMemcpy(result, gpu_out, totalThreads * sizeof(char), 
    hipMemcpyDeviceToHost);
    
    hipEventDestroy(start_time);
    hipEventDestroy(end_time);
    
    /* Turn this block on for verification
    //for (int i=0; i<totalThreads; i++)
    {
        printf("input: %c %d \n", input_text[i], input_text[i]);
        printf("result: %c %d \n", result[i], result[i]);
    }*/    
                                    
    return elapsed;
}
       
int main(int argc, char** argv) {
    // read command line arguments
	int totalThreads = (1 << 20);
    //int totalThreads = 64;                              
	int blockSize = 32;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;
    
	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}
    pageable_main(numBlocks, totalThreads); 
    pinned_main(numBlocks, totalThreads);

	return EXIT_SUCCESS;
}