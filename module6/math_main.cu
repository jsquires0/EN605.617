
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int TOTALTHREADS = 128;
#define THREADS_IN_BLOCK 128

__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

/**
 * Allocates pageable memory for host's input and output arrays
 */
 void MathAlloc( int **pos, int **rnd, 
    int **added, int **subd, 
    int **multd, int **moded)
{
    // allocate
    int *p, *r, *a, *s, *mu, *mo;
    p = (int*)malloc(TOTALTHREADS*sizeof(int));
    r = (int*)malloc(TOTALTHREADS*sizeof(int));
    a = (int*)malloc(TOTALTHREADS*sizeof(int));
    s = (int*)malloc(TOTALTHREADS*sizeof(int));
    mu = (int*)malloc(TOTALTHREADS*sizeof(int));
    mo = (int*)malloc(TOTALTHREADS*sizeof(int));

    // populate input arrays
    for (int i=0; i<TOTALTHREADS; i++)
    {
        p[i] = i;                       
        r[i] = rand() % 4;
    }
    // update pointers                           
    *pos = p;
    *rnd = r;
    *added = a;
    *subd = s;
    *multd = mu;
    *moded = mo;
}

// ******************************** SHARED ******************************* // 
/*
	Adds an array containing integers from 0 to totalThreads
	to an array of random integers between [0,3] and stores the 
	result in output array.
*/
__global__
void sharedAdd(int *pos, int *rnd, int *out)
{
	__shared__ int shared_pos[THREADS_IN_BLOCK];
	__shared__ int shared_rnd[THREADS_IN_BLOCK];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idx_in_block = threadIdx.x;

	shared_pos[idx_in_block] = pos[idx];
	shared_rnd[idx_in_block] = rnd[idx];

	out[idx] = shared_pos[idx_in_block] + shared_rnd[idx_in_block];
}
 
/*
	Subtracts an array of random integers between [0,3] from
    an array containing integers from 0 to totalThreads
	and stores the result in output array.
*/
__global__
void sharedSubtract(int *pos, int *rnd, int *out)
{
	__shared__ int shared_pos[THREADS_IN_BLOCK];
	__shared__ int shared_rnd[THREADS_IN_BLOCK];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idx_in_block = threadIdx.x;

	shared_pos[idx_in_block] = pos[idx];
	shared_rnd[idx_in_block] = rnd[idx];

	out[idx] = shared_pos[idx_in_block] - shared_rnd[idx_in_block];
}
     
/*
	Performs elementwise multiplication of an array of random 
    integers between [0,3] and an array containing integers 
    from 0 to totalThreads, and stores the result in output array
*/
__global__
void sharedMult(int *pos, int *rnd, int *out)
{
	__shared__ int shared_pos[THREADS_IN_BLOCK];
	__shared__ int shared_rnd[THREADS_IN_BLOCK];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idx_in_block = threadIdx.x;

	shared_pos[idx_in_block] = pos[idx];
	shared_rnd[idx_in_block] = rnd[idx];

	out[idx] = shared_pos[idx_in_block] * shared_rnd[idx_in_block];
}
  
/*
	Performs elementwise modular division between 
    an array containing integers from 0 to totalThreads
    and an array of random integers between [0,3].
	Stores the result in output array.
*/
__global__
void sharedMod(int *pos, int *rnd, int *out)
{
	__shared__ int shared_pos[THREADS_IN_BLOCK];
	__shared__ int shared_rnd[THREADS_IN_BLOCK];

	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int idx_in_block = threadIdx.x;

	shared_pos[idx_in_block] = pos[idx];
	shared_rnd[idx_in_block] = rnd[idx];

	out[idx] = shared_pos[idx_in_block] % shared_rnd[idx_in_block];
}

// ******************************** REGISTER ******************************* //   
/*
	Adds an array containing integers from 0 to totalThreads
	to an array of random integers between [0,3] and stores the 
	result in output array.
*/
__global__
void regAdd(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int p_tmp = pos[thread_idx];
	int r_tmp = rnd[thread_idx];
	out[thread_idx] = p_tmp + r_tmp;
}
 
/*
	Subtracts an array of random integers between [0,3] from
    an array containing integers from 0 to totalThreads
	and stores the result in output array.
*/
__global__
void regSubtract(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int p_tmp = pos[thread_idx];
	int r_tmp = rnd[thread_idx];
	out[thread_idx] = p_tmp - r_tmp;
}
      
/*
	Performs elementwise multiplication of an array of random 
    integers between [0,3] and an array containing integers 
    from 0 to totalThreads, and stores the result in output array
*/
__global__
void regMult(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int p_tmp = pos[thread_idx];
	int r_tmp = rnd[thread_idx];
	out[thread_idx] = p_tmp * r_tmp;
}
   
/*
	Performs elementwise modular division between 
    an array containing integers from 0 to totalThreads
    and an array of random integers between [0,3].
	Stores the result in output array.
*/
__global__
void regMod(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int p_tmp = pos[thread_idx];
	int r_tmp = rnd[thread_idx];
	out[thread_idx] = p_tmp % r_tmp;
}
/*
	Execute kernels using register or shared memory
*/
void kernelChoice(int use_reg, int numBlocks, int totalThreads, int *pos,
	int *rnd, int *added, int *subd, int *multd, int *moded)
{
	if (use_reg)
	{
		regAdd<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, added);
		regSubtract<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, subd);
		regMult<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, multd);
		regMod<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, moded);
	}
	else 
	{
		sharedAdd<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, added);
		sharedSubtract<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, subd);
		sharedMult<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, multd);
		sharedMod<<<numBlocks, totalThreads/numBlocks>>>(pos, rnd, moded);
	}
}

/* 
    Calls add, subtract, mult, and mod and performs calculations on gpu
*/
float doMath(int use_reg, int numBlocks, int totalThreads, int *pos, 
                       int *rnd, int *added, int *subd, int *multd, int *moded)
{   
	int *gpu_pos, *gpu_rnd, *gpu_added, *gpu_subd, *gpu_multd, *gpu_moded;
	//allocate gpu memory
	hipMalloc((void**)&gpu_pos, totalThreads * sizeof(int));
	hipMalloc((void**)&gpu_rnd, totalThreads * sizeof(int));
    hipMalloc((void**)&gpu_added, totalThreads * sizeof(int));
    hipMalloc((void**)&gpu_subd, totalThreads * sizeof(int));
    hipMalloc((void**)&gpu_multd, totalThreads * sizeof(int));
    hipMalloc((void**)&gpu_moded, totalThreads * sizeof(int));

	// copy inputs to gpu
	hipMemcpy(gpu_pos, pos, totalThreads * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(gpu_rnd, rnd, totalThreads * sizeof(int),hipMemcpyHostToDevice);
    
	hipEvent_t start_time = get_time();

	// compute results on gpu 
	kernelChoice(use_reg, numBlocks, totalThreads, gpu_pos, gpu_rnd, 
				 gpu_added, gpu_subd, gpu_multd, gpu_moded);

	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	float delta = 0;
	hipEventElapsedTime(&delta, start_time, end_time);

	// copy back to cpu 
	hipMemcpy(added,gpu_added,totalThreads*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(subd, gpu_subd,totalThreads*sizeof(int), hipMemcpyDeviceToHost); 
    hipMemcpy(multd,gpu_multd,totalThreads*sizeof(int),hipMemcpyDeviceToHost); 
    hipMemcpy(moded,gpu_moded,totalThreads*sizeof(int),hipMemcpyDeviceToHost); 
	         
    // clean up                           
	hipFree(gpu_pos); hipFree(gpu_rnd); hipFree(gpu_added); 
	hipFree(gpu_subd); hipFree(gpu_multd); hipFree(gpu_moded); 
	hipEventDestroy(start_time); hipEventDestroy(end_time); hipDeviceReset();
	
    return delta;
}

/* 
    Wrapper to save output to file and print timing metrics
*/
void sub_main(int use_reg, int numBlocks, int totalThreads)
{
    
    int *pos, *rnd, *added, *subd, *multd, *moded;
    MathAlloc(&pos, &rnd, &added, &subd, &multd, &moded);

    // add, subtract, mult, and mod the two input arrays

    float elapsed;
    elapsed = doMath(use_reg, numBlocks, totalThreads, pos, rnd, added, subd, 
						   multd, moded);

	if (use_reg)
	{
		printf("Register memory elapsed: %3.3f ms\n", elapsed); 
	}
	else
	{
		printf("Shared memory elapsed: %3.3f ms\n", elapsed); 
	}

    // Save results
    FILE * outFile;
    outFile = fopen("computed_arrays.txt","w");
    for (int i=0; i<totalThreads; i++)
    {
        fprintf(outFile, "%d\t %d\t %d\t %d\t %d\t %d\t \n", 
                pos[i], rnd[i], added[i], subd[i], multd[i], moded[i]);
    }
    
}

/* 
	Calls all math kernels using either register or shared memory for
	two sets of thread and block sizes.
*/
int main(int argc, char** argv) {

	int numBlocks = TOTALTHREADS/THREADS_IN_BLOCK;
    
	// validate arguments
	if (TOTALTHREADS % THREADS_IN_BLOCK != 0) {

		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("Please update and re-run \n");
	}
	int use_reg = 1;
    // test one
    sub_main(use_reg, numBlocks, TOTALTHREADS);
    sub_main(!use_reg, numBlocks, TOTALTHREADS);

	TOTALTHREADS *= 10;
	numBlocks *= 10;
	
	// test two
	sub_main(use_reg, numBlocks, TOTALTHREADS);
    sub_main(!use_reg, numBlocks, TOTALTHREADS);
	
	return EXIT_SUCCESS;
}