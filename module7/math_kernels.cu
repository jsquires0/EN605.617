#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}

/*
	Adds an array containing integers from 0 to totalThreads
	to an array of random integers between [0,3] and stores the 
	result in output array.
*/
__global__
void add(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	out[thread_idx] = pos[thread_idx] + rnd[thread_idx];

}
  
/*
	Subtracts an array of random integers between [0,3] from
    an array containing integers from 0 to totalThreads
	and stores the result in output array.
*/
__global__
void subtract(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	out[thread_idx] = pos[thread_idx] - rnd[thread_idx];

}
     
/*
	Performs elementwise multiplication of an array of random 
    integers between [0,3] and an array containing integers 
    from 0 to totalThreads, and stores the result in output array
*/
__global__
void mult(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	out[thread_idx] = pos[thread_idx] * rnd[thread_idx];

}

/*
	Performs elementwise modular division between 
    an array containing integers from 0 to totalThreads
    and an array of random integers between [0,3].
	Stores the result in output array.
*/
__global__
void mod(int *pos, int *rnd, int *out)
{
	int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	out[thread_idx] = pos[thread_idx] % rnd[thread_idx];

}

    
/* 
    Calls add, subtract, mult, and mod and performs calculations on gpu
*/
float doMath(hipStream_t stream_0, hipStream_t stream_1,  int totalThreads, int batchSize, int blockSize,
	int *pos, int *rnd, int *added, int *subd, int *multd, int *moded)
{
	int *dev_0_pos, *dev_1_pos, *dev_0_rnd, *dev_1_rnd;
	int	*dev_0_added, *dev_1_added, *dev_0_subd, *dev_1_subd;
	int	*dev_0_multd, *dev_1_multd, *dev_0_moded, *dev_1_moded;
    
    //allocate gpu memory
    hipMalloc((void**)&dev_0_pos, batchSize * sizeof(int));
    hipMalloc((void**)&dev_0_rnd, batchSize * sizeof(int));
    hipMalloc((void**)&dev_0_added, batchSize * sizeof(int));
    hipMalloc((void**)&dev_0_subd, batchSize * sizeof(int));
    hipMalloc((void**)&dev_0_multd, batchSize * sizeof(int));
    hipMalloc((void**)&dev_0_moded, batchSize * sizeof(int));
	
	hipMalloc((void**)&dev_1_pos, batchSize * sizeof(int));
    hipMalloc((void**)&dev_1_rnd, batchSize * sizeof(int));
    hipMalloc((void**)&dev_1_added, batchSize * sizeof(int));
    hipMalloc((void**)&dev_1_subd, batchSize * sizeof(int));
    hipMalloc((void**)&dev_1_multd, batchSize * sizeof(int));
	hipMalloc((void**)&dev_1_moded, batchSize * sizeof(int));
	
    hipEvent_t start_time = get_time();
	// process totalThreads into batches of 16
	for (int i = 0; i<totalThreads, i+= 2*batchSize) {
	
		// copy inputs to GPU
		hipMemcpyAsync(dev_0_pos, pos_i+i, batchSize * sizeof(int), 
					hipMemcpyHostToDevice, stream_0);
		hipMemcpyAsync(dev_1_pos, pos_i+i+batchSize, batchSize * sizeof(int), 
					hipMemcpyHostToDevice, stream_1);		
		hipMemcpyAsync(dev_0_rnd, rnd_i+i, batchSize * sizeof(int), 
					hipMemcpyHostToDevice, stream_0);
		hipMemcpyAsync(dev_1_rnd, rnd_i+i+batchSize, batchSize * sizeof(int), 
					hipMemcpyHostToDevice, stream_1);	

		int numBlocks = batchSize/blockSize;
    	// compute batch results on gpu
		add<<<numBlocks, blockSize, 0, stream_0>>>(dev_0_pos, dev_0_rnd, dev_0_added);
		add<<<numBlocks, blockSize, 0, stream_1>>>(dev_1_pos, dev_1_rnd, dev_1_added);
		subtract<<<numBlocks, blockSize, 0, stream_0>>>(dev_0_pos, dev_0_rnd, dev_0_subd);
		subtract<<<numBlocks, blockSize, 0, stream_1>>>(dev_1_pos, dev_1_rnd, dev_1_subd);
		mult<<<numBlocks, blockSize, 0, stream_0>>>(dev_0_pos, dev_0_rnd, dev_0_multd);
		mult<<<numBlocks, blockSize, 0, stream_1>>>(dev_1_pos, dev_1_rnd, dev_1_multd);
		mod<<<numBlocks, blockSize, 0, stream_0>>>(dev_0_pos, dev_0_rnd, dev_0_moded);
		mod<<<numBlocks, blockSize, 0, stream_1>>>(dev_1_pos, dev_1_rnd, dev_1_moded);
		
		// copy batch back to cpu 
		hipMemcpyAsync(added+i,dev_0_added,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_0);
		hipMemcpyAsync(added+i+batchSize,dev_1_added,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_1);
		hipMemcpyAsync(subd, dev_0_subd,batchSize*sizeof(int), hipMemcpyDeviceToHost, stream_0);
		hipMemcpyAsync(subd+i+batchSize,dev_1_subd,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_1);
		hipMemcpyAsync(multd,dev_0_multd,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_0);
		hipMemcpyAsync(multd+i+batchSize,dev_1_multd,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_1);
		hipMemcpyAsync(moded,dev_0_moded,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_0);
		hipMemcpyAsync(moded+i+batchSize,dev_1_moded,batchSize*sizeof(int),hipMemcpyDeviceToHost, stream_1);
	}
	hipStreamSynchronize(stream_0); hipStreamSynchronize(stream_1);
	hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	float delta = 0;
	hipEventElapsedTime(&delta, start_time, end_time);

    // clean up                           
	hipFree(dev_0_pos); hipFree(dev_0_rnd); hipFree(dev_0_added); 
	hipFree(dev_0_subd); hipFree(dev_0_multd); hipFree(dev_0_moded);
	hipFree(dev_1_pos); hipFree(dev_1_rnd); hipFree(dev_1_added); 
	hipFree(dev_1_subd); hipFree(dev_1_multd); hipFree(dev_1_moded);  
	hipEventDestroy(start_time); hipEventDestroy(end_time); hipDeviceReset();
    return delta;
}