// Computes A*B = C with cuBLAS. A, B, C are NxN matrices

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

int N = 128;

__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}
    
/**
 * Allocates pageable memory for host's input and output arrays
 */
 void pageableMathAlloc(int N, float **A, float **B, float **C)
{
// allocate
float *a, *b, *c;
a = (float*)malloc(N*N*sizeof(float));
b = (float*)malloc(N*N*sizeof(float));
c = (float*)malloc(N*N*sizeof(float));

// update pointers                           
*A = a;
*B = b;
*C = c;
}

/**
 * Allocates pinned memory for hosts input and output arrays
 */
 void pinnedMathAlloc(int N, float **A, float **B, float **C)
{
// allocate
float *a, *b, *c;
hipHostAlloc((void**)&a, N*N*sizeof(float), hipHostMallocDefault);
hipHostAlloc((void**)&b, N*N*sizeof(float), hipHostMallocDefault);           
hipHostAlloc((void**)&c, N*N*sizeof(float), hipHostMallocDefault);  

// update pointers                           
*A = a;
*B = b;
*C = c;
}        

void square_matrix_multiplication(int use_pinned, int N)
{
    // allocate pageable host memory, gpu memory
    float *A, *B, *C, *gpu_A, *gpu_B, *gpu_C;
    use_pinned ? pinnedMathAlloc(N, &A, &B, &C) : pageableMathAlloc(N, &A, &B, &C);
    hipMalloc((void**)&gpu_A, N * N * sizeof(float));
    hipMalloc((void**)&gpu_B, N * N * sizeof(float));
    hipMalloc((void**)&gpu_C, N * N * sizeof(float));

    // use cuRAND to populate A, B on gpu
    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateUniform(rng, gpu_A, N*N);
    hiprandGenerateUniform(rng, gpu_B, N*N);
   
    hipblasHandle_t handle; hipblasCreate(&handle);
    // cuBLAS SGEMM computes (k_1 * A) * B + (k_2 * C)
    float k_1 = 1.0f; float k_2 = 0.0f;
    // start timing of kernel + device -> host transfer
    hipEvent_t start_time = get_time();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &k_1, 
                gpu_A, N, gpu_B, N, &k_2, gpu_C, N);
    
    // copy back to cpu 
	hipMemcpy(A, gpu_A, N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, gpu_B, N*N*sizeof(float), hipMemcpyDeviceToHost); 
    hipMemcpy(C, gpu_C, N*N*sizeof(float), hipMemcpyDeviceToHost); 

    // end timing
    hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	float delta = 0;
    hipEventElapsedTime(&delta, start_time, end_time);
     use_pinned ? printf("Matmul with pinned mem: %3.3f ms\n", delta) : 
                  printf("Matmul with pageable mem: %3.3f ms\n", delta); 

    // cleanup
    hipHostFree(A); hipHostFree(B); hipHostFree(C);
    hipFree(gpu_A); hipFree(gpu_B); hipFree(gpu_C); hipblasDestroy(handle);
    hipEventDestroy(start_time); hipEventDestroy(end_time); hipDeviceReset();
}

/* 
	Computes A*B = C using either pinned or pageable memory for
	two sets matrix sizes
*/
int main(int argc, char** argv) {

	int use_pinned = 1;
    // test one
    square_matrix_multiplication(use_pinned, N);
    square_matrix_multiplication(!use_pinned, N);

	N *= 2;
	// test two
    square_matrix_multiplication(use_pinned, N);
    square_matrix_multiplication(!use_pinned, N);
	
	return EXIT_SUCCESS;
}