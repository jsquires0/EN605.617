// Converts a dense NxN matrix A into CSR format using cuSPARSE
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

int N = 16;
    
__host__ hipEvent_t get_time(void) {
	hipEvent_t time;
	hipEventCreate(&time);
	hipEventRecord(time);
	return time;
}
    
/**
 * Allocates pageable memory for host's input and output arrays
 */
 void pageableNAlloc(int N, float **A, int **row)
{
    // allocate
    float *a; int *r;
    a = (float*)malloc(N*N*sizeof(float));
    r = (int*)malloc((N+1)*sizeof(int));

    // populate input array
    for (int i=0; i< N*N; i++)
    {  
        // 0.5 density
        if (i % 2 == 0){
            a[i] = 1.0f; 
        }
        else {
            a[i] = 0.0f;
        }                      
    }

    // update pointers                           
    *A = a;
    *row = r;
}
                     
/**
 * Allocates pageable memory for host's input and output arrays
 */
 void pageableNNZAlloc(int nnz, float **val, int **col)
{
    // allocate
    float *v; int *c;
    v = (float*)malloc(nnz*sizeof(float));
    c = (int*)malloc(nnz*sizeof(int));

    // update pointers                           
    *col = c;
    *val = v;
}

/**
 * Allocates pinned memory for hosts input and output arrays
 */
 void pinnedNAlloc(int N, float **A, int **row)
{
    // allocate
    float *a; int *r;
    hipHostAlloc((void**)&a, N*N*sizeof(float), hipHostMallocDefault); 
    hipHostAlloc((void**)&r, (N+1)*sizeof(int), hipHostMallocDefault);

    // populate input array
    for (int i=0; i< N*N; i++)
    {  
        // 0.5 density
        if (i % 2 == 0){
            a[i] = 1.0f; 
        }
        else {
            a[i] = 0.0f;
        }                      
    }
    // update pointers                           
    *A = a;
    *row = r;
}
/**
 * Allocates pageable memory for host's input and output arrays
 */
 void pinnedNNZAlloc(int nnz, float **val, int **col)
{
    // allocate
    float *v; int *c;
    hipHostAlloc((void**)&v, nnz*sizeof(float), hipHostMallocDefault); 
    hipHostAlloc((void**)&c, nnz*sizeof(int), hipHostMallocDefault);

    // update pointers                           
    *col = c;
    *val = v;
}

int64_t do_conversion(hipsparseHandle_t handle, hipsparseDnMatDescr_t dense_mat, 
    hipsparseSpMatDescr_t sparse_mat, int N, float *gpu_DnA, float **gpu_val, 
    int **gpu_col, int **gpu_row)
{
    // Three steps to convert gpu_DnA -> gpu_SpA (csr format)
    // First, determine size of workspace buffer:
    size_t memBuff = 0, *gpu_memBuff;
    hipsparseDenseToSparse_bufferSize(handle, dense_mat, sparse_mat, 
            HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &memBuff);
    hipMalloc((void**)&gpu_memBuff, memBuff);

    // Second, determine the number of non zero (nnz) elements in A. This is 
    // necessary because we haven't yet allocated device memory for gpu_col and 
    // gpu_val because their size is equal to the number of non zero (nnz) 
    // elements of A. Though we know nnz = N/2, to keep code flexible, get it:
    hipsparseDenseToSparse_analysis(handle, dense_mat, sparse_mat,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, gpu_memBuff);
    
    int64_t tmp_1, tmp_2, nnz;
    float *tmp_val; int*tmp_col, *tmp_row;
    hipsparseSpMatGetSize(sparse_mat, &tmp_1, &tmp_2, &nnz);
    hipMalloc((void**)&tmp_val, nnz * sizeof(float));
    hipMalloc((void**)&tmp_col, nnz * sizeof(int));
    hipMalloc((void**)&tmp_row, nnz * sizeof(int));

    // Finally, perform the dense -> csr conversion
    // and set gpu_val, gpu_col, gpu_row to point to the result,
    hipsparseCsrSetPointers(sparse_mat, tmp_row, tmp_col, tmp_val);
    hipsparseDenseToSparse_convert(handle, dense_mat, sparse_mat,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, gpu_memBuff);

    *gpu_row = tmp_row; 
    *gpu_col = tmp_col;
    *gpu_val = tmp_val;

    return nnz;
}

void dense_to_csr(int use_pinned, int N){
    // allocate pageable host memory, gpu memory
    float *DnA, *gpu_DnA, *gpu_val;
    int *row, *gpu_row, *gpu_col;
    use_pinned ? pinnedNAlloc(N, &DnA, &row) : pageableNAlloc(N, &DnA, &row);
    hipMalloc((void**)&gpu_DnA, N * N * sizeof(float));
    hipMalloc((void**)&gpu_row, (N+1) * sizeof(int));

    // copy A, B host -> device and start timing of kernels
    hipMemcpy(gpu_DnA, DnA, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipsparseHandle_t handle; hipsparseCreate(&handle);
    hipEvent_t start_time = get_time();
    
    // create matrices
    // signature: https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-generic-dnmat-create
    hipsparseDnMatDescr_t dense_mat;
    hipsparseCreateDnMat(&dense_mat, N, N, N, gpu_DnA, HIP_R_32F, HIPSPARSE_ORDER_ROW);
    // signature: https://docs.nvidia.com/cuda/cusparse/index.html#cusparse-generic-spmat-create-csr
    hipsparseSpMatDescr_t sparse_mat;
    hipsparseCreateCsr(&sparse_mat, N, N, 0, gpu_row, NULL, NULL, HIPSPARSE_INDEX_32I, 
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    int64_t nnz;
    nnz = do_conversion(handle, dense_mat, sparse_mat, N, gpu_DnA, &gpu_val, 
                        &gpu_col, &gpu_row);
    // copy back to cpu
    int *col; float *val;
    use_pinned ? pinnedNNZAlloc(nnz, &val, &col) : pageableNAlloc(nnz, &val, &col);
	hipMemcpy(row, gpu_row, (N+1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col, gpu_col, nnz * sizeof(int), hipMemcpyDeviceToHost); 
    hipMemcpy(val, gpu_val, nnz * sizeof(float), hipMemcpyDeviceToHost); 
    // end timing
    hipEvent_t end_time = get_time();
	hipEventSynchronize(end_time);
	float delta = 0;
    hipEventElapsedTime(&delta, start_time, end_time);
    use_pinned ? printf("Dense -> Sparse (CSR) with pinned mem: %3.3f ms\n", delta)
               : printf("Dense -> Sparse (CSR) with pageable mem: %3.3f ms\n", delta);
    // cleanup
    hipHostFree(DnA); hipHostFree(row); hipHostFree(col); hipHostFree(val);
    hipFree(gpu_DnA); hipFree(gpu_row); hipFree(gpu_col); hipFree(gpu_val);
    hipEventDestroy(start_time); hipEventDestroy(end_time); hipDeviceReset();
    hipsparseDestroy(handle); hipsparseDestroyDnMat(dense_mat); 
    hipsparseDestroySpMat(sparse_mat);
}

/* 
    Converts a half empty square matrix into sparse CSR format using either 
    pinned or pageable memory for two sets of matrix sizes
*/
int main(int argc, char** argv) {

	int use_pinned = 1;
    // test one
    dense_to_csr(use_pinned, N);
    dense_to_csr(!use_pinned, N);

	N *= 2;
	// test two
    dense_to_csr(use_pinned, N);
    dense_to_csr(!use_pinned, N);
	
	return EXIT_SUCCESS;
}